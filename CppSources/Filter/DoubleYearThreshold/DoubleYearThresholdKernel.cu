#include "hip/hip_runtime.h"
#pragma once
#include "../../Generator/HomoPoly/CUDA/HomoPolyMethod.cu"


const int __NUM_THRESHOLD_PER_CYCLE__ = 5;


__device__ double max_of_array(double *array, int left, int right, double supremum){
    double max_ = __NEGATIVE_INFINITY__;
    for (int i=left; i<right; i++){
        if (array[i] < supremum && array[i] > max_) max_ = array[i];
    }
    return max_;
}


__device__ void top_n_of_array(double *array, int left, int right, double *result, int start, int n){
    double supremum = __POSITIVE_INFINITY__;
    for (int i=0; i<n; i++){
        supremum = max_of_array(array, left, right, supremum);
        result[start+i] = supremum;
    }
}


__global__ void fill_thresholds(double *weights, double *thresholds, int *INDEX, int index_length, int num_array, int length){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int num_cycle = index_length - 2;
    if (index < num_array*num_cycle){
        int ix = index % num_cycle;
        int iy = index / num_cycle;
        top_n_of_array(weights + iy*length,
                       INDEX[ix+1], INDEX[ix+2],
                       thresholds + iy*__NUM_THRESHOLD_PER_CYCLE__*num_cycle,
                       ix*__NUM_THRESHOLD_PER_CYCLE__, __NUM_THRESHOLD_PER_CYCLE__);
    }
}


__device__ void _double_year_threshold_investing(double *weight, double threshold, int t_idx, double *result,
    double INTEREST, int *INDEX, double *PROFIT, int *SYMBOL, int *BOOL_ARG, int index_size, int num_cycle){
    int reason = 0;
    double Geo2 = 0, Har2 = 0;
    int start, end, end2, count, k, sym, s, rs_idx;
    double temp, n;
    bool check;
    for (int i=index_size-3; i>0; i--){
        start = INDEX[i];
        end = INDEX[i+1];
        temp = 0;
        count = 0;
        check = false;
        if (!reason){
            end2 = INDEX[i+2];
            for (k=start; k<end; k++){
                if (weight[k] > threshold && BOOL_ARG[k]){
                    check = true;
                    sym = SYMBOL[k];
                    for (s=end; s<end2; s++){
                        if (SYMBOL[s] == sym){
                            if (weight[s] > threshold){
                                count++;
                                temp += PROFIT[k];
                            }
                            break;
                        }
                    }
                }
            }
        } else {
            for (k=start; k<end; k++){
                if (weight[k] > threshold && BOOL_ARG[k]){
                    check = true;
                    count++;
                    temp += PROFIT[k];
                }
            }
        }

        if (!count){
            Geo2 += log(INTEREST);
            Har2 += 1.0 / INTEREST;
            if (!check) reason = 1;
        } else {
            temp /= count;
            Geo2 += log(temp);
            Har2 += 1.0 / temp;
            reason = 0;
        }

        if (i <= num_cycle && t_idx+1 >= i){
            rs_idx = num_cycle - i;
            n = index_size - 2 - i;
            result[2*rs_idx] = exp(Geo2/n);
            result[2*rs_idx+1] = n / Har2;
        }
    }
}


__global__ void double_year_threshold_investing(double *weights, double *thresholds, double *results, int num_array, int num_threshold,
    int length, int num_cycle, double INTEREST, int *INDEX, double *PROFIT, int *SYMBOL, int *BOOL_ARG, int index_size){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_array*num_threshold){
        int ix = index % num_threshold;
        int iy = index / num_threshold;
        _double_year_threshold_investing(
            weights + iy*length,
            thresholds[iy*num_threshold + ix],
            ix / __NUM_THRESHOLD_PER_CYCLE__,
            results + iy*num_threshold*num_cycle*2 + ix*num_cycle*2,
            INTEREST, INDEX, PROFIT, SYMBOL, BOOL_ARG, index_size, num_cycle
        );
    }
}


__global__ void find_best_results(double *results, double *thresholds, double *finals, int num_array, int num_threshold, int num_cycle){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < 2*num_array*num_cycle){
        int iz = index % 2;
        int ix = (index/2) % num_cycle;
        int iy = (index/2) / num_cycle;

        double *result = results + iy*num_threshold*num_cycle*2;
        double *threshold = thresholds + iy*num_threshold;
        double *final_ = finals + iy*num_cycle*4 + ix*4;

        final_[2*iz] = threshold[0];
        final_[2*iz + 1] = result[2*ix + iz];
        for (int i=1; i<num_threshold; i++){
            if (result[i*num_cycle*2 + 2*ix + iz] > final_[2*iz + 1]){
                final_[2*iz] = threshold[i];
                final_[2*iz + 1] = result[i*num_cycle*2 + 2*ix + iz];
            }
        }
    }
}

#pragma once
#include <hip/hip_runtime.h>



__device__ __forceinline__ void merge(
    double *keys,
    double *vals,
    int left,
    int mid,
    int right,
    double *temp_keys,
    double *temp_vals
) {
    int i = left, j = mid + 1;
    double nextKey, nextVal;
    int k = 0;

    while (i <= mid && j <= right){
        if (keys[i] >= keys[j]){
            nextKey = keys[i];
            nextVal = vals[i];
            i++;
        }
        else {
            nextKey = keys[j];
            nextVal = vals[j];
            j++;
        }
        temp_keys[left+k] = nextKey;
        temp_vals[left+k] = nextVal;
        k++;
    }

    while (i <= mid){
        temp_keys[left+k] = keys[i];
        temp_vals[left+k] = vals[i];
        i++; k++;
    }

    while (j <= right){
        temp_keys[left+k] = keys[j];
        temp_vals[left+k] = vals[j];
        j++; k++;
    }

    for (i=left; i<=right; i++){
        keys[i] = temp_keys[i];
        vals[i] = temp_vals[i];
    }
}


__device__ __forceinline__ void mergeSort(
    double *keys,
    double *vals,
    int n,
    double *temp_keys,
    double *temp_vals
) {
    int curr_size;
    int left, mid, right;

    for (curr_size=1; curr_size<n; curr_size*=2){
        for (left=0; left<n-1; left+=2*curr_size){
            if (left+curr_size-1 < n-1) mid = left + curr_size - 1;
            else mid = n - 1;

            if (left+2*curr_size-1 < n-1) right = left + 2*curr_size - 1;
            else right = n - 1;

            merge(keys, vals, left, mid, right, temp_keys, temp_vals);
        }
    }
}


__device__ __forceinline__ double mean(
    double *array,
    int start,
    int end
) {
    double temp = 0.0;
    for (int i=start; i<end; i++) temp += array[i];

    return temp / (end - start);
}


__device__ __forceinline__ double noBalance_point(
    double *sortedProfit,
    int n
) {
    int cur_idx = n / 2;
    int i;

    for (i=cur_idx; i>0; i--){
        if (mean(sortedProfit, 0, i) <= mean(sortedProfit, i, n)){
            if (i == cur_idx) return 100000000.0;
            return i + 1.0;
        }
    }

    return 1.0;
}


__device__ __forceinline__ void _get_noBalance_value(
    double *weight,
    double *profit,
    double *temp_wgt,
    double *temp_prf,
    int *INDEX,
    int index_size,
    int num_cycle,
    double *result
) {
    int i, start, end, n, rs_idx;
    double temp_sum = 0.0;
    int count = 0;

    for (i=index_size-2; i>0; i--){
        start = INDEX[i];
        end = INDEX[i+1];
        n = end - start;

        mergeSort(weight + start, profit + start, n, temp_wgt + start, temp_prf + start);

        temp_sum += noBalance_point(profit + start, n);
        count++;

        if (i <= num_cycle){
            rs_idx = num_cycle - i;
            if (temp_sum >= 1e8) result[rs_idx] = -1e8;
            else result[rs_idx] = -temp_sum / count;
        }
    }
}


__global__ void get_noBalance_value(
    double *weights,
    double *profits,
    double *temp_wgts,
    double *temp_prfs,
    int length,
    int *INDEX,
    int index_size,
    int num_cycle,
    double *results,
    int num_array
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_array){
        _get_noBalance_value(
            weights + index*length,
            profits + index*length,
            temp_wgts + index*length,
            temp_prfs + index*length,
            INDEX,
            index_size,
            num_cycle,
            results + index*num_cycle
        );
    }
}

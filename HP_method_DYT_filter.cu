#include "CppSources/Filter/DoubleYearThreshold/DoubleYearThresholdFilter.cu"


int main(int argc, char *argv[]){
    if (argc == 1){
        raise_error("Command chua nhap config_path", ".\\ExeFile\\HP_method_DYT_filter_CUDA.exe <config_path>");
    }

    string config_path = argv[1];
    DoubleYearThresholdFilter vis(config_path);
    vis.run();
}
